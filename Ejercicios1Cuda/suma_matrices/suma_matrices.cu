
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <sys/time.h>

const int N=20000;
double cpuSecond()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}

__global__ void MatAdd( float *A, float *B, float *C, int N)
{
int j = blockIdx.x * blockDim.x + threadIdx.x;  // Compute row index
int i = blockIdx.y * blockDim.y + threadIdx.y;  // Compute column index
int index=i*N+j; // Compute global 1D index
if (i < N && j < N)
	C[index] = A[index] + B[index]; // Compute C element
}

int main()
{
int i;
const int NN=N*N;
/* pointers to host memory */
/* Allocate arrays A, B and C on host*/
float * A = (float*) malloc(NN*sizeof(float));
float * B = (float*) malloc(NN*sizeof(float));
float * C = (float*) malloc(NN*sizeof(float));

/* pointers to device memory */
float *A_d, *B_d, *C_d;
/* Allocate arrays a_d, b_d and c_d on device*/
hipMalloc ((void **) &A_d, sizeof(float)*NN);
hipMalloc ((void **) &B_d, sizeof(float)*NN);
hipMalloc ((void **) &C_d, sizeof(float)*NN);

/* Initialize arrays a and b */
for (i=0; i<NN;i++)
{
  A[i]= (float) i;
  B[i]= -(float) i;
}

double  t1 = cpuSecond();
/* Copy data from host memory to device memory */
hipMemcpy(A_d, A, sizeof(float)*NN, hipMemcpyHostToDevice);
hipMemcpy(B_d, B, sizeof(float)*NN, hipMemcpyHostToDevice);

/* Compute the execution configuration */
dim3 threadsPerBlock (32, 32);
dim3 numBlocks( ceil ((float)(N)/threadsPerBlock.x), ceil ((float)(N)/threadsPerBlock.y) );
MatAdd <<<numBlocks, threadsPerBlock>>> (A_d, B_d, C_d, N);


/* Copy data from deveice memory to host memory */
hipMemcpy(C, C_d, sizeof(float)*NN, hipMemcpyDeviceToHost);

double Tgpu = cpuSecond()-t1;
printf("%d=%f\n",N,Tgpu);


/* Free the memory */
free(A); free(B); free(C);
hipFree(A_d); hipFree(B_d);hipFree(C_d);

}
