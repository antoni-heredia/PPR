
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <sys/time.h>

double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

__global__ void calcularPI(double *A, double step, int N )
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute row index
  if (i < N)
  {
    double x = (i + 1 - 0.5) * step;
    A[i] = 4.0 / (1.0 + x * x);
  }
}
__global__ void reduceSum(double *d_V, int n)
{
  extern __shared__ double sdata[];

  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
  double suma = (i < n) ? d_V[i] : 0;
  if (i + blockDim.x < n) 
    suma += d_V[i + blockDim.x];
  sdata[tid] = suma;
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      sdata[tid] = suma += sdata[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0)
    d_V[blockIdx.x] = suma;
}
int main()
{

  // Calculo de pi secuencial
  static long num_steps = 1000000;
  double step;

  double t1 = cpuSecond();
  double x, pi, sum = 0.0;
  step = 1.0 / (double)num_steps;
  for (int i = 1; i <= num_steps; i++)
  {
    x = (i - 0.5) * step;
    sum = sum + 4.0 / (1.0 + x * x);
  }
  pi = step * sum;
  double Tcpu = cpuSecond() - t1;
  printf("El resultado de pi secuencial es: %f\n",pi);
  printf("El tiempo secuencial=%f\n", Tcpu);

  /* pointers to host memory */
  /* Allocate arrays A, B and C on host*/
  double *A = (double *)malloc(num_steps * sizeof(double));

  /* pointers to device memory */
  double *A_d;
  /* Allocate arrays a_d, b_d and c_d on device*/
  hipMalloc((void **)&A_d, sizeof(double) * num_steps);

  t1 = cpuSecond();

  /* Compute the execution configuration */
  int threadsPerBlock = 1024;
  int numBlocks = ceil(((float)num_steps) / threadsPerBlock);
  calcularPI<<<numBlocks, threadsPerBlock>>>(A_d, step, num_steps);

  /* Copy data from deveice memory to host memory */
  hipMemcpy(A, A_d, sizeof(double) * num_steps, hipMemcpyDeviceToHost);
  pi = 0;
  for(int i = 0; i < num_steps;i++)
    pi+=A[i];
  pi = pi*step;
  double Tgpu = cpuSecond() - t1;
  printf("El resultado de pi paralelo es: %f\n",pi);
  printf("Tiempo de CPU=%f\n", Tgpu);
  /* Free the memory */
  free(A);
  hipFree(A_d);
}
