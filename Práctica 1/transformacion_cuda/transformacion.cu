
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>

// CUDA runtime
//#include <cuda_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <helper_cuda.h>

using namespace std;

__global__ void transformacion(float *A, float *B, float *C,int n){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < n){

    int inicio = blockIdx.x * blockDim.x;
    int final = inicio+blockDim.x;
    float suma = 0;
    for(int x = inicio; x < final; x++){
      if(x<n){
        float aux = A[x]*i;
        suma += aux;
        suma += ((int)ceil(aux) % 2 == 0) ? B[x] : -B[x];

      }
    }
    C[i] = suma;
  }
}

__global__ void transformacion_compartida(float *A, float *B, float *C,int n){

  extern __shared__ float sdata[]; 
  float *sA = sdata; 	   
  float *sB = sdata+blockDim.x;    
  
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if(i < n){
    sA[threadIdx.x] = A[i]; 
    sB[threadIdx.x] = B[i];
    __syncthreads();

    float suma = 0;
    for(int x = 0; x < blockDim.x; x++){
      if(x<n){
        float aux = sA[x]*i;
        suma += aux;
        suma += ((int)ceil(aux) % 2 == 0) ? sB[x] : -sB[x];
      }
    }
    C[i] = suma;
  }
}

__global__ void suma_bloque(float *C,float*D,int n){

  extern __shared__ float sdata[];
  int tid = threadIdx.x;
  int i = blockIdx.x *blockDim.x  + threadIdx.x;
  sdata[tid] = ((i < n) ? C[i] : 0.0f);

  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0){
    D[blockIdx.x] = sdata[tid];
  }

}
__global__ void mayor(float * C,float * E, int N){
  extern __shared__ float sdata[];
  int tid = threadIdx.x;
  int i = blockIdx.x *blockDim.x  + threadIdx.x;
  sdata[tid] = ((i < N) ? C[i] : 0.0f);
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      sdata[tid] = (sdata[tid + s] > sdata[tid]) ? sdata[tid+s] : sdata[tid];
    }
    __syncthreads();
  }

  if (tid == 0){
    E[blockIdx.x] = sdata[tid];
  }

}
//**************************************************************************
int main(int argc, char *argv[])
//**************************************************************************
{
  int Bsize, NBlocks;
  if (argc != 3)
  {
    cout << "Uso: transformacion Num_bloques Tam_bloque  " << endl;
    return (0);
  }
  else
  {
    NBlocks = atoi(argv[1]);
    Bsize = atoi(argv[2]);
  }

  const int N = Bsize * NBlocks;
  //* pointers to host memory */

  float *A, *B, *C, *D, *E;
  float *A_device, *B_device, *C_device, * D_device, *E_device;
  //* Allocate arrays a, b and c on host*/
  A = new float[N];
  B = new float[N];
  C = new float[N];
  D = new float[NBlocks];
  E = new float[NBlocks];

  int size = N*sizeof(float);
  int size_d = NBlocks*sizeof(float);
  hipError_t err;

  err = hipMalloc((void **) &A_device, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA" << endl;
  }
  err = hipMalloc((void **) &B_device, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA" << endl;
  }
  err = hipMalloc((void **) &C_device, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA" << endl;
  }
  err = hipMalloc((void **) &D_device,size_d);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA" << endl;
  }
  err = hipMalloc((void **) &E_device,size_d);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA" << endl;
  }
  //float mx; // maximum of C

  //* Initialize arrays A and B */
  for (int i = 0; i < N; i++)
  {
    A[i] = (float)(1 - (i % 100) * 0.001);
    B[i] = (float)(0.5 + (i % 10) * 0.1);
    //A[i] = 0;
    //B[i] = 1;
  }

  // Time measurement
  double t1 = clock();
  //Copio los datos de host a device
  err = hipMemcpy(A_device, A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A GPU" << endl;
  }
  err = hipMemcpy(B_device, B, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A GPU" << endl;
  }

  int blockSize = Bsize;
  int blockNum = NBlocks;

  transformacion<<<blockNum,blockSize, blockSize*2*sizeof(float)>>>(A_device,B_device,C_device, N);
  suma_bloque<<<blockNum,blockSize, blockSize*sizeof(float)>>>(C_device,D_device,N);
  mayor<<<blockNum,blockSize, blockSize*sizeof(float)>>>(C_device,E_device,N);

  err = hipGetLastError();

  if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch transformacion shared kernel!\n");
      cout << err << endl;
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(D, D_device, size_d, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A GPU C" << endl;
  }

  err = hipMemcpy(E, E_device, size_d, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A GPU C" << endl;
  }

  double t2 = clock();
  t2 = (t2 - t1) / CLOCKS_PER_SEC;
  
  for(int c = 0; c < NBlocks ; c++){
    cout << "D[" << c << "]=" << D[c] << endl;
  }

  float mayor = 0;
  for(int c = 0; c < NBlocks ; c++){
    mayor = (mayor > E[c]) ? mayor : E[c];
  }
  cout << "El valor máximo en C es:  " << mayor << endl;
  cout << "N=" << N << "= " << Bsize << "*" << NBlocks << "  ........  Tiempo gastado CPU= " << t2 << endl
  << endl;  /*
  //for (int i=0; i<N;i++)   cout<<"C["<<i<<"]="<<C[i]<<endl;
  cout << "................................." << endl;
  for (int k = 0; k < NBlocks; k++)
    cout << "D[" << k << "]=" << D[k] << endl;
  //cout << "................................." << endl
  //     << "El valor máximo en C es:  " << mx << endl;

  cout << endl
       << "N=" << N << "= " << Bsize << "*" << NBlocks << "  ........  Tiempo gastado CPU= " << t2 << endl
       << endl;
*/
  //* Free the memory */
  delete (A);
  delete (B);
  delete (C);
  delete (D);
  delete (E);

}