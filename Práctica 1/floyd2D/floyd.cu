#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string.h>
#include <sys/time.h>
#include "Graph.h"

// CUDA runtime
//#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

#define blocksize 1024

using namespace std;

//**************************************************************************
double cpuSecond()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}
// Version 1D
//**************************************************************************
__global__ void floyd_kernel(int * M, const int nverts, const int k) {
	int ij = threadIdx.x + blockDim.x * blockIdx.x;
	if (ij < nverts * nverts) {
		int Mij = M[ij];
		int i= ij / nverts;
		int j= ij - i * nverts;
		if (i != j && i != k && j != k) {
				int Mikj = M[i * nverts + k] + M[k * nverts + j];
			Mij = (Mij > Mikj) ? Mikj : Mij;
			M[ij] = Mij;
			}
	}
}
// Versión 2D
__global__ void floyd_kernel_2D(int * M, const int nverts, const int k) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	
	if ( i < nverts && j < nverts ) {
		int indice = j + nverts * i; 
		int Mindice = M[indice];
   		if (i != j && i != k && j != k) {
			int Mikj = M[i * nverts + k] + M[k * nverts + j];
			Mindice = (Mindice > Mikj) ? Mikj : Mindice;
			M[indice] = Mindice;
		}
	}
}
__global__ void reduceSum(int *d_V, int n)
{
  extern __shared__ int sdata[blocksize];

  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

	  if (i < n){ 
    
		sdata[tid] = d_V[i];

		__syncthreads();

		for (int s = blockDim.x; s > 0; s >>= 1)
		{
			if (tid < s)
			{
			sdata[tid] = (sdata[tid] > sdata[tid+s] ? sdata[tid] : sdata[tid+s]);
			}
			__syncthreads();
		}
	}
  if (tid == 0){
	d_V[blockIdx.x] = sdata[0];
  }
}

int main (int argc, char *argv[]) {

	if (argc != 2) {
		cerr << "Sintaxis: " << argv[0] << " <archivo de grafo>" << endl;
		return(-1);
	}
	

  //Get GPU information
  int devID;
  hipDeviceProp_t props;
  hipError_t err;
  err = hipGetDevice(&devID);
  if(err != hipSuccess) {
		cout << "ERRORRR" << endl;
	}


hipGetDeviceProperties(&props, devID);
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

	Graph G;
	G.lee(argv[1]);// Read the Graph

	//cout << "EL Grafo de entrada es:"<<endl;
	//G.imprime();
	const int nverts = G.vertices;
	const int niters = nverts;

	const int nverts2 = nverts * nverts;

	int *c_Out_M = new int[nverts2];
	int size = nverts2*sizeof(int);
	int * d_In_M = NULL;

	err = hipMalloc((void **) &d_In_M, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA" << endl;
	}

	int *A = G.Get_Matrix();

	// GPU phase
	double  t1 = cpuSecond();

	err = hipMemcpy(d_In_M, A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A GPU" << endl;
	}
	int threadsPerBlock = blocksize;
	int blocksPerGrid = (nverts2 + threadsPerBlock - 1) / threadsPerBlock;
	for(int k = 0; k < niters; k++) {
		//printf("CUDA kernel launch \n");
	 	

	  floyd_kernel<<<blocksPerGrid,threadsPerBlock >>>(d_In_M, nverts, k);
	  err = hipGetLastError();

	  if (err != hipSuccess) {
	  	fprintf(stderr, "Failed to launch kernel! ERROR= %d\n",err);
	  	exit(EXIT_FAILURE);
		}
	}

	hipMemcpy(c_Out_M, d_In_M, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	double Tgpu = cpuSecond()-t1;

	cout << "Tiempo gastado GPU= " << Tgpu << endl;


	//Ejecución del kernel 2D
	int *B = G.Get_Matrix();
	int *c_Out_M_2D = new int[nverts2];
	int * d_In_M_2D = NULL;
	
	err = hipMalloc((void **) &d_In_M_2D, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA" << endl;
	}
	
	t1 = cpuSecond();
	err = hipMemcpy(d_In_M_2D, B, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A GPU" << endl;
	}

	dim3 threads_2D (32, 32);
	dim3 blocks_2D( ceil ((float)(nverts)/threads_2D.x), ceil ((float)(nverts)/threads_2D.y) );
	for(int k=0; k < niters; k++) {
	 	
		floyd_kernel_2D<<<blocks_2D,threads_2D >>>(d_In_M_2D, nverts, k);

		err = hipGetLastError();

		if (err != hipSuccess) {
			fprintf(stderr, "Failed to launch kernel! ERROR= %d\n",err);
			exit(EXIT_FAILURE);
		}
	}
	hipMemcpy(c_Out_M_2D, d_In_M_2D, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	double Tgpu_2D = cpuSecond()-t1;

	cout << "Tiempo gastado GPU en 2D= " << Tgpu_2D << endl;
	// CPU phase
	t1 = cpuSecond();

	// BUCLE PPAL DEL ALGORITMO
	int inj, in, kn;
	for(int k = 0; k < niters; k++) {
          kn = k * nverts;
	  for(int i=0;i<nverts;i++) {
			in = i * nverts;
			for(int j = 0; j < nverts; j++)
	       			if (i!=j && i!=k && j!=k){
			 	    inj = in + j;
			 	    A[inj] = min(A[in+k] + A[kn+j], A[inj]);
	       }
	   }
	}

	double t2 = cpuSecond() - t1;
	cout << "Tiempo gastado CPU= " << t2 << endl;
	cout << "Ganancia GPU_1d sobre CPU= " << t2 / Tgpu << endl;
	cout << "Ganancia GPU_2d SOBRE CPU=" << t2 / Tgpu_2D << endl;
	cout << "Ganancia GPU_1d SOBRE GPU_2D=" << Tgpu / Tgpu_2D << endl;
		
	hipMemcpy(d_In_M_2D,c_Out_M_2D , size, hipMemcpyHostToDevice);
	int bloquesR = ceil(float(nverts2)/blocksize);

	reduceSum<<<bloquesR,blocksize>>>(d_In_M_2D,nverts2);
	hipMemcpy(c_Out_M_2D,d_In_M_2D , size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
 
	int longitud = c_Out_M_2D[0];
	for(int i = 1; i < bloquesR;i++){
		longitud=(c_Out_M_2D[i] > longitud ? c_Out_M_2D[i]:longitud);
	}

	cout << "La longitud del camino es: " << longitud << endl;


	for(int i = 0; i < nverts; i++)
		for(int j = 0;j < nverts; j++)
			if (abs(c_Out_M[i*nverts+j] - G.arista(i,j)) > 0)
				cout << "Error (" << i << "," << j << ")   " << c_Out_M[i*nverts+j] << "..." << G.arista(i,j) << endl;

}
